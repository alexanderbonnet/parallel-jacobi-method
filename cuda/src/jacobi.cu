#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 2048
#define THREADS_PER_BLOCK 128
#define N_BLOCKS (N / THREADS_PER_BLOCK + 1)
#define ELEMENT double

ELEMENT mean(ELEMENT *x_array, int size) {
    ELEMENT mean = 0;
    for (unsigned int i = 0; i < size; i++) {
        mean += x_array[i];
    }
    mean = mean / size;
    return mean;
}

ELEMENT *init_x(unsigned int size, ELEMENT value) {
    ELEMENT *x_vect = (ELEMENT *)malloc(sizeof(ELEMENT) * size);
    for (unsigned int i = 0; i < size; i++) {
        x_vect[i] = value;
    }
    return x_vect;
}

ELEMENT *init_a(unsigned int size) {
    ELEMENT *a_matrix = (ELEMENT *)malloc(sizeof(ELEMENT) * size * size);
    for (unsigned int i = 0; i < size; i++) {
        for (unsigned int j = 0; j < size; j++) {
            if (j == i) {
                a_matrix[i + j * size] = 2 * size + 1;
            } else {
                a_matrix[i + j * size] = 1;
            }
        }
    }
    return a_matrix;
}

__global__ void warm_up_gpu() {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    ELEMENT ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid;
}

__global__ void criterion(ELEMENT *a, ELEMENT *b, ELEMENT *crit) {
    __shared__ ELEMENT temp[THREADS_PER_BLOCK];

    *crit = 0;
    unsigned int index = threadIdx.x;
    ELEMENT sum = 0;
    ELEMENT diff = 0;
    for (unsigned int i = index; i < N; i += THREADS_PER_BLOCK) {
        diff = a[i] - b[i];
        sum += diff * diff;
    }
    temp[index] = sum;

    __syncthreads();
    for (unsigned int size = THREADS_PER_BLOCK / 2; size > 0; size /= 2) {
        if (index < size) {
            temp[index] += temp[index + size];
        }
        __syncthreads();
    }
    if (index == 0) {
        *crit = temp[0];
    }
}

__global__ void increment_x(ELEMENT *x_new, ELEMENT *x_old, ELEMENT *a_mat,
                            ELEMENT *b_vec) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int indexN = N * index;

    if (index < N) {
        ELEMENT sum = 0;
        for (unsigned int j = 0; j < N; j++) {
            if (index != j) {
                sum += a_mat[indexN + j] * x_old[j];
            }
        }
        x_new[index] = (b_vec[index] - sum) / a_mat[indexN + index];
    }
}

unsigned int solve_with_jacobi(ELEMENT *x_init, ELEMENT *a_mat, ELEMENT *b_vec,
                               ELEMENT epsilon) {
    unsigned int nit = 0;
    ELEMENT eps_2 = epsilon * epsilon;
    ELEMENT crit = eps_2 + 1;

    ELEMENT *dev_a, *dev_b, *dev_x_old, *dev_x_new, *dev_crit;

    hipMalloc((void **)&dev_a, N * N * sizeof(ELEMENT));
    hipMalloc((void **)&dev_b, N * sizeof(ELEMENT));
    hipMalloc((void **)&dev_x_old, N * sizeof(ELEMENT));
    hipMalloc((void **)&dev_x_new, N * sizeof(ELEMENT));
    hipMalloc((void **)&dev_crit, sizeof(ELEMENT));

    // copy inputs to device
    hipMemcpy(dev_a, a_mat, N * N * sizeof(ELEMENT), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b_vec, N * sizeof(ELEMENT), hipMemcpyHostToDevice);
    hipMemcpy(dev_x_old, x_init, N * sizeof(ELEMENT), hipMemcpyHostToDevice);
    hipMemcpy(dev_x_new, x_init, N * sizeof(ELEMENT), hipMemcpyHostToDevice);
    hipMemcpy(dev_crit, &crit, sizeof(ELEMENT), hipMemcpyHostToDevice);

    while (crit > eps_2) {
        increment_x<<<N_BLOCKS, THREADS_PER_BLOCK>>>(dev_x_new, dev_x_old,
                                                     dev_a, dev_b);
        criterion<<<1, THREADS_PER_BLOCK>>>(dev_x_new, dev_x_old, dev_crit);
        hipMemcpy(dev_x_old, dev_x_new, N * sizeof(ELEMENT),
                   hipMemcpyDeviceToDevice);
        hipMemcpy(&crit, dev_crit, sizeof(ELEMENT), hipMemcpyDeviceToHost);
        nit += 1;
    }
    hipMemcpy(x_init, dev_x_new, N * sizeof(ELEMENT), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_x_old);
    hipFree(dev_x_new);
    hipFree(dev_crit);

    return nit;
}

int main(int argc, char *argv[]) {
    unsigned int num_executions = 20;

    unsigned int nit = 0;
    ELEMENT result = 0;

    ELEMENT eps = 1e-6;

    ELEMENT *a_mat = init_a(N);
    ELEMENT *x_init = init_x(N, 1);
    ELEMENT *b_vec = init_x(N, 6);

    ELEMENT *execution_times =
        (ELEMENT *)malloc(sizeof(ELEMENT) * num_executions);

    struct timeval t1, t2;
    ELEMENT time = 0;

    printf(
        "running iterative Jacobi algorithm with size = "
        "%d\n",
        N);

    warm_up_gpu<<<N / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>();

    for (unsigned int i = 0; i < num_executions; i++) {
        ELEMENT *x_solve = (ELEMENT *)malloc(N * sizeof(ELEMENT));
        memcpy(x_solve, x_init, N * sizeof(ELEMENT));

        gettimeofday(&t1, 0);

        nit = solve_with_jacobi(x_solve, a_mat, b_vec, eps);

        hipDeviceSynchronize();
        gettimeofday(&t2, 0);

        time = 1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec;
        time = time / 1000.0;

        execution_times[i] = time;
        printf("exec time = %f \n", time);

        result = x_solve[0];

        free(x_solve);
    }

    ELEMENT avg_time = mean(execution_times, num_executions);

    printf("number of iterations = %d \n", nit);
    printf("execution time = %.10fs \n", avg_time);
    printf("\n");
    printf("our result = %.10f \n", result);
    printf("theoretical result = %.10f \n", 2.0 / N);

    free(execution_times);
    free(a_mat);
    free(b_vec);
    free(x_init);

    return 0;
}
